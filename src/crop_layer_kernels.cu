#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "crop_layer.h"
#include "utils.h"
#include "hip/hip_runtime.h"
#include "image.h"
}

__device__ real get_pixel_kernel(real *image, int w, int h, int x, int y, int c)
{
    if(x < 0 || x >= w || y < 0 || y >= h) return 0;
    return image[x + w*(y + c*h)];
}

__device__ real3 rgb_to_hsv_kernel(real3 rgb)
{
    real r = rgb.x;
    real g = rgb.y; 
    real b = rgb.z;

    real h, s, v;
    real max = (r > g) ? ( (r > b) ? r : b) : ( (g > b) ? g : b);
    real min = (r < g) ? ( (r < b) ? r : b) : ( (g < b) ? g : b);
    real delta = max - min;
    v = max;
    if(max == 0){
        s = 0;
        h = -1;
    }else{
        s = delta/max;
        if(r == max){
            h = (g - b) / delta;
        } else if (g == max) {
            h = 2 + (b - r) / delta;
        } else {
            h = 4 + (r - g) / delta;
        }
        if (h < 0) h += 6;
    }
#if REAL == DOUBLE
    return make_double3(h, s, v);
#else
    return make_float3(h, s, v);
#endif
}

__device__ real3 hsv_to_rgb_kernel(real3 hsv)
{
    real h = hsv.x;
    real s = hsv.y; 
    real v = hsv.z;

    real r, g, b;
    real f, p, q, t;

    if (s == 0) {
        r = g = b = v;
    } else {
        int index = (int) floorf(h);
        f = h - index;
        p = v*(1-s);
        q = v*(1-s*f);
        t = v*(1-s*(1-f));
        if(index == 0){
            r = v; g = t; b = p;
        } else if(index == 1){
            r = q; g = v; b = p;
        } else if(index == 2){
            r = p; g = v; b = t;
        } else if(index == 3){
            r = p; g = q; b = v;
        } else if(index == 4){
            r = t; g = p; b = v;
        } else {
            r = v; g = p; b = q;
        }
    }
    r = (r < 0) ? 0 : ((r > 1) ? 1 : r);
    g = (g < 0) ? 0 : ((g > 1) ? 1 : g);
    b = (b < 0) ? 0 : ((b > 1) ? 1 : b);
#if REAL == DOUBLE
    return make_double3(r, g, b);
#else
    return make_float3(r, g, b);
#endif
}

__device__ real bilinear_interpolate_kernel(real *image, int w, int h, real x, real y, int c)
{
    int ix = (int) floorf(x);
    int iy = (int) floorf(y);

    real dx = x - ix;
    real dy = y - iy;

    real val = (1-dy) * (1-dx) * get_pixel_kernel(image, w, h, ix, iy, c) + 
        dy     * (1-dx) * get_pixel_kernel(image, w, h, ix, iy+1, c) + 
        (1-dy) *   dx   * get_pixel_kernel(image, w, h, ix+1, iy, c) +
        dy     *   dx   * get_pixel_kernel(image, w, h, ix+1, iy+1, c);
    return val;
}

__global__ void levels_image_kernel(real *image, real *rand, int batch, int w, int h, int train, real saturation, real exposure, real translate, real scale, real shift)
{
    int size = batch * w * h;
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= size) return;
    int x = id % w;
    id /= w;
    int y = id % h;
    id /= h;
    real rshift = rand[0];
    real gshift = rand[1];
    real bshift = rand[2];
    real r0 = rand[8*id + 0];
    real r1 = rand[8*id + 1];
    real r2 = rand[8*id + 2];
    real r3 = rand[8*id + 3];

    saturation = r0*(saturation - 1) + 1;
    saturation = (r1 > .5f) ? 1.f/saturation : saturation;
    exposure = r2*(exposure - 1) + 1;
    exposure = (r3 > .5f) ? 1.f/exposure : exposure;

    size_t offset = id * h * w * 3;
    image += offset;
    real r = image[x + w*(y + h*0)];
    real g = image[x + w*(y + h*1)];
    real b = image[x + w*(y + h*2)];
#if REAL == DOUBLE
    real3 rgb = make_double3(r,g,b);
#else
    real3 rgb = make_float3(r,g,b);
#endif
    if(train){
        real3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }
    image[x + w*(y + h*0)] = rgb.x*scale + translate + (rshift - .5f)*shift;
    image[x + w*(y + h*1)] = rgb.y*scale + translate + (gshift - .5f)*shift;
    image[x + w*(y + h*2)] = rgb.z*scale + translate + (bshift - .5f)*shift;
}

__global__ void forward_crop_layer_kernel(real *input, real *rand, int size, int c, int h, int w, int crop_height, int crop_width, int train, int flip, real angle, real *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= size) return;

    real cx = w/2.f;
    real cy = h/2.f;

    int count = id;
    int j = id % crop_width;
    id /= crop_width;
    int i = id % crop_height;
    id /= crop_height;
    int k = id % c;
    id /= c;
    int b = id;

    real r4 = rand[8*b + 4];
    real r5 = rand[8*b + 5];
    real r6 = rand[8*b + 6];
    real r7 = rand[8*b + 7];

    real dw = (w - crop_width)*r4;
    real dh = (h - crop_height)*r5;
    flip = (flip && (r6 > .5f));
    angle = 2*angle*r7 - angle;
    if(!train){
        dw = (w - crop_width)/2.f;
        dh = (h - crop_height)/2.f;
        flip = 0;
        angle = 0;
    }

    input += w*h*c*b;

    real x = (flip) ? w - dw - j - 1 : j + dw;    
    real y = i + dh;

    real rx = cosf(angle)*(x-cx) - sinf(angle)*(y-cy) + cx;
    real ry = sinf(angle)*(x-cx) + cosf(angle)*(y-cy) + cy;

    output[count] = bilinear_interpolate_kernel(input, w, h, rx, ry, k);
}

extern "C" void forward_crop_layer_gpu(crop_layer layer, network net)
{
    cuda_random(layer.rand_gpu, layer.batch*8);

    real radians = layer.angle*3.14159265f/180.f;

    real scale = 2;
    real translate = -1;
    if(layer.noadjust){
        scale = 1;
        translate = 0;
    }

    int size = layer.batch * layer.w * layer.h;

    levels_image_kernel<<<cuda_gridsize(size), BLOCK>>>(net.input_gpu, layer.rand_gpu, layer.batch, layer.w, layer.h, net.train, layer.saturation, layer.exposure, translate, scale, layer.shift);
    check_error(hipPeekAtLastError());

    size = layer.batch*layer.c*layer.out_w*layer.out_h;

    forward_crop_layer_kernel<<<cuda_gridsize(size), BLOCK>>>(net.input_gpu, layer.rand_gpu, size, layer.c, layer.h, layer.w, layer.out_h, layer.out_w, net.train, layer.flip, radians, layer.output_gpu);
    check_error(hipPeekAtLastError());

/*
       cuda_pull_array(layer.output_gpu, layer.output, size);
       image im = real_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 0*(size/layer.batch));
       image im2 = real_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 1*(size/layer.batch));
       image im3 = real_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 2*(size/layer.batch));

       translate_image(im, -translate);
       scale_image(im, 1/scale);
       translate_image(im2, -translate);
       scale_image(im2, 1/scale);
       translate_image(im3, -translate);
       scale_image(im3, 1/scale);
       
       show_image(im, "cropped");
       show_image(im2, "cropped2");
       show_image(im3, "cropped3");
       cvWaitKey(0);
       */
}


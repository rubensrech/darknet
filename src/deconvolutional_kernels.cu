#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

// extern "C" {
#include "convolutional_layer.h"
#include "deconvolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
// }

void forward_deconvolutional_layer_gpu(layer l, network net)
{
    int i;

    int m = l.size*l.size*l.n;
    int n = l.h*l.w;
    int k = l.c;

    fill_gpu(l.outputs*l.batch, CAST(0), l.output_gpu, 1);

    for(i = 0; i < l.batch; ++i){
        real *a = l.weights_gpu;
        real *b = net.input_gpu + i*l.c*l.h*l.w;
        real *c = net.workspace;

        gemm_gpu(1,0,m,n,k,CAST(1),a,m,b,n,CAST(0),c,n);

        col2im_gpu(net.workspace, l.out_c, l.out_h, l.out_w, l.size, l.stride, l.pad, l.output_gpu+i*l.outputs);
    }
    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, net);
    } else {
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
    activate_array_gpu(l.output_gpu, l.batch*l.n*l.out_w*l.out_h, l.activation);
}

void backward_deconvolutional_layer_gpu(layer l, network net)
{
    int i;

    //constrain_gpu(l.outputs*l.batch, 1, l.delta_gpu, 1);
    gradient_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);

    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, net);
    } else {
        backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }

    //if(net.delta_gpu) memset(net.delta_gpu, 0, l.batch*l.h*l.w*l.c*sizeof(real));

    for(i = 0; i < l.batch; ++i){
        int m = l.c;
        int n = l.size*l.size*l.n;
        int k = l.h*l.w;

        real *a = net.input_gpu + i*m*k;
        real *b = net.workspace;
        real *c = l.weight_updates_gpu;

        im2col_gpu(l.delta_gpu + i*l.outputs, l.out_c, l.out_h, l.out_w, 
                l.size, l.stride, l.pad, b);
        gemm_gpu(0,1,m,n,k,CAST(1),a,k,b,k,CAST(1),c,n);

        if(net.delta_gpu){
            int m = l.c;
            int n = l.h*l.w;
            int k = l.size*l.size*l.n;

            real *a = l.weights_gpu;
            real *b = net.workspace;
            real *c = net.delta_gpu + i*n*m;

            gemm_gpu(0,0,m,n,k,CAST(1),a,k,b,n,CAST(1),c,n);
        }
    }
}

void pull_deconvolutional_layer(layer l)
{
    cuda_pull_array(l.weights_gpu, l.weights, l.c*l.n*l.size*l.size);
    cuda_pull_array(l.biases_gpu, l.biases, l.n);
    cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.c*l.n*l.size*l.size);
    cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_pull_array(l.scales_gpu, l.scales, l.n);
        cuda_pull_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_pull_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

void push_deconvolutional_layer(layer l)
{
    cuda_push_array(l.weights_gpu, l.weights, l.c*l.n*l.size*l.size);
    cuda_push_array(l.biases_gpu, l.biases, l.n);
    cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.c*l.n*l.size*l.size);
    cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_push_array(l.scales_gpu, l.scales, l.n);
        cuda_push_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_push_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

void update_deconvolutional_layer_gpu(layer l, update_args a)
{
    real learning_rate = a.learning_rate*l.learning_rate_scale;
    real momentum = a.momentum;
    real decay = a.decay;
    int batch = a.batch;

    if(a.adam){
        adam_update_gpu(l.weights_gpu, l.weight_updates_gpu, l.m_gpu, l.v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.nweights, batch, a.t);
        adam_update_gpu(l.biases_gpu, l.bias_updates_gpu, l.bias_m_gpu, l.bias_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        if(l.scales_gpu){
            adam_update_gpu(l.scales_gpu, l.scale_updates_gpu, l.scale_m_gpu, l.scale_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        }
    }else{
        axpy_gpu(l.nweights, CAST(-decay*batch), l.weights_gpu, 1, l.weight_updates_gpu, 1);
        axpy_gpu(l.nweights, CAST(learning_rate/batch), l.weight_updates_gpu, 1, l.weights_gpu, 1);
        scal_gpu(l.nweights, momentum, l.weight_updates_gpu, 1);

        axpy_gpu(l.n, CAST(learning_rate/batch), l.bias_updates_gpu, 1, l.biases_gpu, 1);
        scal_gpu(l.n, momentum, l.bias_updates_gpu, 1);

        if(l.scales_gpu){
            axpy_gpu(l.n, CAST(learning_rate/batch), l.scale_updates_gpu, 1, l.scales_gpu, 1);
            scal_gpu(l.n, momentum, l.scale_updates_gpu, 1);
        }
    }
}


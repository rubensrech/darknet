#include "hip/hip_runtime.h"
#include "real.h"

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

template<typename T1, typename T2>
__global__ void array_cast_kernel(T1 *src, T2 *dst, int n) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		dst[i] = src[i];
	}
}

void half2real_array_gpu(half_host* src, real* dst, int n) {
	array_cast_kernel<<<cuda_gridsize(n), BLOCK>>>((half_device*)src, (real_device*)dst, n);
	check_error(hipPeekAtLastError());
}
void real2half_array_gpu(real* src, half_host* dst, int n) {
	array_cast_kernel<<<cuda_gridsize(n), BLOCK>>>((real_device*)src, (half_device*)dst, n);
	check_error(hipPeekAtLastError());
}

void float2real_array_gpu(float* src, real* dst, int n) {
	array_cast_kernel<<<cuda_gridsize(n), BLOCK>>>(src, (real_device*)dst, n);
	check_error(hipPeekAtLastError());
}

void real2float_array_gpu(real* src, float* dst, int n) {
	array_cast_kernel<<<cuda_gridsize(n), BLOCK>>>((real_device*)src, dst, n);
	check_error(hipPeekAtLastError());
}


template<typename T1, typename T2>
void generic_copy_array_gpu_template(T1 *src, T2 *dst, int n) {
	array_cast_kernel<<<cuda_gridsize(n), BLOCK>>>(src, dst, n);
	check_error(hipPeekAtLastError());
}
void generic_copy_array_gpu(half_host *src, half_host *dst, int n) {
	generic_copy_array_gpu_template((half_device*)src, (half_device*)dst, n);
}
void generic_copy_array_gpu(half_host *src, float *dst, int n) {
	generic_copy_array_gpu_template((half_device*)src, dst, n);
}
void generic_copy_array_gpu(float *src, half_host *dst, int n) {
	generic_copy_array_gpu_template(src, (half_device*)dst, n);
}
void generic_copy_array_gpu(float *src, float *dst, int n) {
	generic_copy_array_gpu_template(src, dst, n);
}
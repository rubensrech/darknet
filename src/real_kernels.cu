#include "hip/hip_runtime.h"
#include "real.h"

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

template<typename T1, typename T2>
__global__ void array_cast_kernel(T1* src, T2* dst, int n) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		dst[i] = src[i];
	}
}

void half2real_array_gpu(half_host* src, real* dst, int n) {
	array_cast_kernel<<<cuda_gridsize(n), BLOCK>>>((half_device*)src, (real_device*)dst, n);
	check_error(hipPeekAtLastError());
}
void real2half_array_gpu(real* src, half_host* dst, int n) {
	array_cast_kernel<<<cuda_gridsize(n), BLOCK>>>((real_device*)src, (half_device*)dst, n);
	check_error(hipPeekAtLastError());
}

void float2real_array_gpu(float* src, real* dst, int n) {
	array_cast_kernel<<<cuda_gridsize(n), BLOCK>>>(src, (real_device*)dst, n);
	check_error(hipPeekAtLastError());
}

void real2float_array_gpu(real* src, float* dst, int n) {
	array_cast_kernel<<<cuda_gridsize(n), BLOCK>>>((real_device*)src, dst, n);
	check_error(hipPeekAtLastError());
}